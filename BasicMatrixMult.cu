#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<math.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include <stdlib.h>
#include "GpuTimer.h"
using namespace std;

#define BLOCK_SIZE 16


//Compute C=A*B
// Serial implementation for running on CPU using a single thread.
void MatrixMultiplyCpu(float* A, float* B, float* C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
	//@@ Insert Your Code Here for the CPU Function to Compute Matrix Maltiply
	for (int row=0;row<numARows;row++)
		for (int col=0;col<numBColumns;col++)
		{
			float sum=0;
			for (int ii = 0; ii < numAColumns; ii++) {
				sum += A[row * numAColumns + ii] * B[ii * numBColumns + col];
			}
			C[row * numBColumns + col] = sum;
		}
}


//GPU Kernel for Basic Matrix Multiplication
__global__ void BasixMatrixMultiply(float* A, float* B, float* C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
	//@@ Insert Your Code Here for the CUDA Kernel for Basic Matrix Multiply
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < numARows && col < numBColumns) {
		float sum = 0;
		for (int ii = 0; ii < numAColumns; ii++) {
			sum += A[row * numAColumns + ii] * B[ii * numBColumns + col];
		}
		C[row * numBColumns + col] = sum;
	}

}



int main(void)
{
	
	int numARows=960; // number of rows in the matrix A
	int numAColumns=640; // number of columns in the matrix A
	int numBRows=640; // number of rows in the matrix B
	int numBColumns=800; // number of columns in the matrix B
	
	int numCRows; // number of rows in the matrix C (you have to set this)
	int numCColumns; // number of columns in the matrix C (you have to set this)
	
	//@@ Insert Your Code Here to Set numCRows and numCColumns
	numCRows=numARows;
	numCColumns=numBColumns;

	//Allocate the host memory for the input and output matrices
	float *h_A = (float *)malloc(sizeof(float)*numARows*numAColumns);
	float *h_B = (float *)malloc(sizeof(float)*numBRows*numBColumns);
	float *h_C = (float *)malloc(sizeof(float)*numCRows*numCColumns);
	float *h_C_CPU = (float *)malloc(sizeof(float)*numCRows*numCColumns);


	//Random Initialize Matrix A. 
	//There are several ways to do this, such as making functions for manual input or using random numbers. 
	//In this case, we simply use a for loop to fill the cells with trigonometric values of the indices:
	// Set the Seed for the random number generator rand() 
	//srand(clock());
	for (int i=0; i<numARows; i++)
	{
		for (int j=0; j<numAColumns; j++)
		{
			h_A[i*numAColumns+j]=(float)rand()*4.0 /(float)(RAND_MAX);
			//h_A[i*numAColumns+j]=sin(i);
		}
	}

	//Random Initialize Matrix B
	for (int i=0; i<numBRows; i++)
	{
		for (int j=0; j<numBColumns; j++)
		{
			h_B[i*numBColumns+j]=(float)rand()*4.0 /(float)(RAND_MAX) ;
			//h_B[i*numBColumns+j]=cos(j);

		}
	}


	

	

	//Allocate memory on the device for input and output matrices and record the needed time
	float *d_A, *d_B, *d_C;
	GpuTimer timer;
	timer.Start();

	//@@Insert Your Code Here to allocate memory for d_A, d_B, d_C
	hipMalloc((void **)&d_A, numARows * numAColumns * sizeof(float));
	hipMalloc((void **)&d_B,numBRows * numBColumns * sizeof(float));
	hipMalloc((void **)&d_C,numARows * numBColumns * sizeof(float));
	
	timer.Stop();
	printf("Time to allocate memory on the device is: %f msecs.\n", timer.Elapsed());



	//Copy the input matrices A and B from the host to the device and record the needed time
	GpuTimer timer1;
	timer1.Start();
	
	//@@ Insert Your Code Here to copy matrices A and B from Host to Device
	hipMemcpy(d_A, h_A, numARows * numAColumns * sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, numBRows * numBColumns * sizeof(float),hipMemcpyHostToDevice);

	timer1.Stop();
	printf("Time to copy the Matrix from the host to the device is: %f msecs.\n", timer1.Elapsed());


	//Do the Processing on the GPU
	//@@ Insert Kernel Execution Configuration Parameters
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 dimGrid(ceil(((float)numCColumns) / BLOCK_SIZE),ceil(((float)numCRows) / BLOCK_SIZE));
	hipMemset(d_C, 0, numARows * numBColumns * sizeof(float));
	//Invoke the BasicMatrixMultiply kernel and record the needed time for its execution
	GpuTimer timer2;
	timer2.Start();
	//@@ Insert Your Code Here for Kernel Invocation
	BasixMatrixMultiply <<<dimGrid, dimBlock>>> (d_A, d_B, d_C, numARows,numAColumns, numBRows, numBColumns, numCRows, numCColumns);
	timer2.Stop();
	printf("Implemented CUDA code ran in: %f msecs.\n", timer2.Elapsed());

	//Copy resulting matrix from device to host and record the needed time
	GpuTimer timer3;
	timer3.Start();
	//@@ Insert Your Code Here to Copy the resulting Matrix d_C from device to the Host h_C
	hipMemcpy(h_C, d_C, numARows * numBColumns * sizeof(float),hipMemcpyDeviceToHost);
	timer3.Stop();
	printf("Time to copy the resulting Matrix from the device to the host is: %f msecs.\n", timer3.Elapsed());


	//Do the Processing on the CPU
	clock_t begin = clock();
	//@@ Insert Your Code Here to call the CPU function MatrixMultiplyCpu where the resulting matrix is h_C_CPU
	MatrixMultiplyCpu(h_A, h_B, h_C_CPU, numARows,numAColumns, numBRows, numBColumns, numCRows, numCColumns);
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
	printf("Implemented CPU serial code ran in: %f msecs.\n", time_spent);

	//Verify Results Computed by GPU and CPU
	for (int i=0; i<numCRows; i++)
		for (int j=0; j<numCColumns; j++)
		
			if (fabs(h_C_CPU[i*numCColumns+j] - h_C[i*numCColumns+j]) > 1e-2)
			{
				fprintf(stderr, "Result verification failed at element (%d,%d) !\n", i,j);
				exit(EXIT_FAILURE);
			}
	printf("Test PASSED\n");


	//Free host memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_CPU);

	//Free device memory
	//@@ Insert Your Code Here to Free Device Memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;

}
